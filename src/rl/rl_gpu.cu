#include "hip/hip_runtime.h"
#include <thrust/execution_policy.h>
#include <thrust/scan.h>

#include "rl_gpu.cuh"
#include "../utils.cuh"

namespace RunLength
{
    // Main functions

    RLCompressed gpuCompress(uint8_t *data, size_t size)
    {
        if (size == 0)
        {
            return RLCompressed{
                .outputValues = nullptr,
                .outputCounts = nullptr,
                .count = 0};
        }

        // Copy input data to GPU
        uint8_t *d_data;
        CHECK_CUDA(hipMalloc(&d_data, sizeof(uint8_t) * size));
        CHECK_CUDA(hipMemcpy(d_data, data, sizeof(uint8_t) * size, hipMemcpyHostToDevice));

        // Prepare GPU arrays
        uint32_t *d_startMask;
        CHECK_CUDA(hipMalloc(&d_startMask, sizeof(uint32_t) * size));
        CHECK_CUDA(hipMemset(d_startMask, 0, sizeof(uint32_t) * size));
        uint32_t *d_scannedStartMask;
        CHECK_CUDA(hipMalloc(&d_scannedStartMask, sizeof(uint32_t) * size));
        uint32_t *d_startIndices;
        CHECK_CUDA(hipMalloc(&d_startIndices, sizeof(uint32_t) * size));
        uint32_t *d_startIndicesLength;
        CHECK_CUDA(hipMalloc(&d_startIndicesLength, sizeof(uint32_t)));
        // We could do it only after we know how much exactly we need, but it doesn't really matter
        // as we will copy back exact amount back to cpu anyway.
        // This way error handling is easier as all allocations are done at the beggining of the function.
        uint8_t *d_outputValues;
        CHECK_CUDA(hipMalloc(&d_outputValues, sizeof(uint8_t) * size));
        uint8_t *d_outputCounts;
        CHECK_CUDA(hipMalloc(&d_outputCounts, sizeof(uint8_t) * size));
        // Same here, we could wait and allocate it later with exact size, but this way it's easier
        // to handle errors.
        uint32_t *d_recalculateSequence;
        CHECK_CUDA(hipMalloc(&d_recalculateSequence, sizeof(uint32_t) * size));
        uint32_t *d_shouldRecalculate;
        CHECK_CUDA(hipMalloc(&d_shouldRecalculate, sizeof(uint32_t)));
        CHECK_CUDA(hipMemset(d_shouldRecalculate, 0, sizeof(uint32_t)));

        // Calculate start mask
        const uint32_t calculateStartMaskThreadsCount = 1024;
        const uint32_t calculateStartMaskBlocksCount = ceil(size * 1.0 / calculateStartMaskThreadsCount);
        compressCalculateStartMask<<<calculateStartMaskBlocksCount, calculateStartMaskThreadsCount>>>(d_data, size, d_startMask);
        CHECK_CUDA(hipDeviceSynchronize());
        CHECK_CUDA(hipGetLastError());

        // Calculate scanned start mask
        compressCalculateScannedStartMask(d_startMask, d_scannedStartMask, size);

        // Calculate start indicies
        const uint32_t calculateStartIndiciesThreadsCount = 1024;
        const uint32_t calculateStartIndiciesBlocksCount = ceil(size * 1.0 / calculateStartIndiciesThreadsCount);
        compressCalculateStartIndicies<<<calculateStartIndiciesBlocksCount, calculateStartIndiciesThreadsCount>>>(d_scannedStartMask, size, d_startIndices, d_startIndicesLength);
        CHECK_CUDA(hipDeviceSynchronize());
        CHECK_CUDA(hipGetLastError());

        // First copy to CPU size of final output to know how much bytes to copy (and allocate)
        // and to know how big kernel should be
        uint32_t outputSize = 0;
        CHECK_CUDA(hipMemcpy(&outputSize, d_startIndicesLength, sizeof(uint32_t), hipMemcpyDeviceToHost));

        // Check if we need to recalculate some sequence due to size > 255
        const uint32_t checkForMoreSequencesThreadsCount = 1024;
        const uint32_t checkForMoreSequencesBlocksCount = ceil(outputSize * 1.0 / checkForMoreSequencesThreadsCount);
        compressCheckForMoreSequences<<<checkForMoreSequencesBlocksCount, checkForMoreSequencesThreadsCount>>>(d_startIndices, d_startIndicesLength, size, d_recalculateSequence, d_shouldRecalculate);
        CHECK_CUDA(hipDeviceSynchronize());
        CHECK_CUDA(hipGetLastError());

        // Copy to cpu boolean value to check if need to recalculate some sequences
        uint32_t shouldRecalculate = 0;
        CHECK_CUDA(hipMemcpy(&shouldRecalculate, d_shouldRecalculate, sizeof(uint32_t), hipMemcpyDeviceToHost));

        // FIXME: this part doesnt work
        if (shouldRecalculate != 0)
        {
            printf("here\n");

            // Copy data to CPU needed for threads counts of next kernel
            uint32_t lastRecalculateSequence;
            CHECK_CUDA(hipMemcpy(&lastRecalculateSequence, &d_recalculateSequence[outputSize - 1], sizeof(uint32_t), hipMemcpyDeviceToHost));

            // Prescan on `recalculateSequence`
            compressRecalculateSequencePrescan(d_recalculateSequence, outputSize);

            // Copy data to CPU needed for threads counts of next kernel
            uint32_t lastRecalculateSequencePrescan;
            CHECK_CUDA(hipMemcpy(&lastRecalculateSequencePrescan, &d_recalculateSequence[outputSize - 1], sizeof(uint32_t), hipMemcpyDeviceToHost));

            // Recalculate start mask
            const uint32_t recalculateStartMaskAllThreads = lastRecalculateSequence + lastRecalculateSequencePrescan;
            const uint32_t recalculateStartMaskThreadsCount = 1024;
            const uint32_t recalculateStartMaskBlocksCount = ceil(recalculateStartMaskAllThreads * 1.0 / recalculateStartMaskThreadsCount);
            compressRecalculateStartMask<<<recalculateStartMaskBlocksCount, recalculateStartMaskThreadsCount>>>(d_startMask, recalculateStartMaskAllThreads, d_recalculateSequence, outputSize, d_startIndices);
            CHECK_CUDA(hipDeviceSynchronize());
            CHECK_CUDA(hipGetLastError());

            // Do again points 2. and 3.
            // Calculate scanned start mask
            compressCalculateScannedStartMask(d_startMask, d_scannedStartMask, size);

            // Calculate start indicies
            compressCalculateStartIndicies<<<calculateStartIndiciesBlocksCount, calculateStartIndiciesThreadsCount>>>(d_scannedStartMask, size, d_startIndices, d_startIndicesLength);
            CHECK_CUDA(hipDeviceSynchronize());
            CHECK_CUDA(hipGetLastError());

            // Copy to CPU final outputSize
            uint32_t outputSize = 0;
            CHECK_CUDA(hipMemcpy(&outputSize, d_startIndicesLength, sizeof(uint32_t), hipMemcpyDeviceToHost));
        }

        // Calculate final output
        const uint32_t calculateOutputThreadsCount = 1024;
        const uint32_t calculateOutputBlocksCount = ceil(outputSize * 1.0 / calculateOutputThreadsCount);
        compressCalculateOutput<<<calculateOutputBlocksCount, calculateOutputThreadsCount>>>(d_data, size, d_startIndices, d_startIndicesLength, d_outputValues, d_outputCounts);
        CHECK_CUDA(hipDeviceSynchronize());
        CHECK_CUDA(hipGetLastError());

        // Allocate needed cpu arrays
        uint8_t *outputValues = reinterpret_cast<uint8_t *>(malloc(sizeof(uint8_t) * outputSize));
        if (outputValues == nullptr)
        {
            throw std::runtime_error("Cannot allocate memory");
        }
        uint8_t *outputCounts = reinterpret_cast<uint8_t *>(malloc(sizeof(uint8_t) * outputSize));
        if (outputCounts == nullptr)
        {
            throw std::runtime_error("Cannot allocate memory");
        }

        // Copy results to CPU
        CHECK_CUDA(hipMemcpy(outputValues, d_outputValues, sizeof(uint8_t) * outputSize, hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(outputCounts, d_outputCounts, sizeof(uint8_t) * outputSize, hipMemcpyDeviceToHost));

        // Deallocate GPU arrays
        hipFree(d_data);
        hipFree(d_startMask);
        hipFree(d_scannedStartMask);
        hipFree(d_startIndices);
        hipFree(d_startIndicesLength);
        hipFree(d_outputValues);
        hipFree(d_outputCounts);

        return RLCompressed{
            .outputValues = outputValues,
            .outputCounts = outputCounts,
            .count = outputSize,
        };
    }

    RLDecompressed gpuDecompress(uint8_t *values, uint8_t *counts, size_t size)
    {
        // TODO:
    }

    // Kernels

    __global__ void compressCalculateStartMask(uint8_t *d_data, size_t size, uint32_t *d_startMask)
    {
        auto threadId = blockDim.x * blockIdx.x + threadIdx.x;
        if (threadId == 0 || (threadId > 0 && threadId < size && d_data[threadId] != d_data[threadId - 1]))
        {
            d_startMask[threadId] = 1;
        }
    }

    __global__ void compressCalculateStartIndicies(uint32_t *d_scannedStartMask, size_t size, uint32_t *d_startIndicies, uint32_t *d_startIndiciesLength)
    {
        __shared__ uint32_t s_maxLength[1];
        auto threadId = blockDim.x * blockIdx.x + threadIdx.x;
        auto localThreadId = threadIdx.x;

        // Initialize shared memory
        if (localThreadId == 0)
        {
            // It will always be at least 1, in case of length 0 we early return from main compress function
            s_maxLength[0] = 1;
        }
        __syncthreads();

        if (threadId == 0)
        {
            d_startIndicies[0] = 0;
        }
        else if (threadId < size && d_scannedStartMask[threadId] != d_scannedStartMask[threadId - 1])
        {
            auto id = d_scannedStartMask[threadId] - 1;
            d_startIndicies[id] = threadId;
            // + 1 because we want the length, not the index
            atomicMax(&s_maxLength[0], id + 1);
        }
        __syncthreads();

        // Save currently biggest changed index in global variable
        if (localThreadId == 0)
        {
            atomicMax(d_startIndiciesLength, s_maxLength[0]);
        }
    }

    __global__ void compressCheckForMoreSequences(uint32_t *d_startIndicies, uint32_t *d_startIndiciesLength, size_t size, uint32_t *d_recalculateSequence, uint32_t *d_shouldRecalculate)
    {
        __shared__ uint32_t s_shouldRecalculate[1];
        __shared__ uint32_t s_startIndiciesLength[1];
        auto threadId = blockDim.x * blockIdx.x + threadIdx.x;
        auto localThreadId = threadIdx.x;

        // Initialize shared memory
        if (localThreadId == 0)
        {
            s_shouldRecalculate[0] = false;
            s_startIndiciesLength[0] = d_startIndiciesLength[0];
        }
        __syncthreads();

        // Case when there is only one sequence
        if (s_startIndiciesLength[0] == 1)
        {
            if (threadId == 0)
            {
                uint32_t diff = size;
                if (diff > 255)
                {
                    d_recalculateSequence[0] = diff / 255;
                    atomicOr(s_shouldRecalculate, 1);
                }
            }
        }
        else if (threadId < s_startIndiciesLength[0] - 1)
        {
            auto diff = d_startIndicies[threadId + 1] - d_startIndicies[threadId];
            if (diff > 255)
            {
                d_recalculateSequence[threadId] = diff / 255;
                atomicOr(s_shouldRecalculate, 1);
            }
        }
        __syncthreads();

        // Save result from shared to global memory
        if (localThreadId == 0)
        {
            atomicOr(d_shouldRecalculate, s_shouldRecalculate[0]);
        }
    }

    __global__ void compressCalculateOutput(uint8_t *d_data, size_t size, uint32_t *d_startIndicies, uint32_t *d_startIndiciesLength, uint8_t *d_outputValues, uint8_t *d_outputCounts)
    {
        __shared__ uint32_t s_length[1];
        auto threadId = blockDim.x * blockIdx.x + threadIdx.x;
        auto localThreadId = threadIdx.x;

        // Initialize shared memory
        if (localThreadId == 0)
        {
            s_length[0] = d_startIndiciesLength[0];
        }
        __syncthreads();

        if (threadId < s_length[0])
        {
            d_outputValues[threadId] = d_data[d_startIndicies[threadId]];
        }

        if (threadId == s_length[0] - 1)
        {
            d_outputCounts[threadId] = (uint8_t)((uint32_t)size - d_startIndicies[threadId]);
        }
        else if (threadId < s_length[0] - 1)
        {
            d_outputCounts[threadId] = d_startIndicies[threadId + 1] - d_startIndicies[threadId];
        }
    }

    __global__ void compressRecalculateStartMask(uint32_t *d_startMask, uint32_t allThreads, uint32_t *d_recalculateSequence, size_t recalculateSequenceLength, uint32_t *d_startIndicies)
    {
        auto threadId = blockDim.x * blockIdx.x + threadIdx.x;
        if (threadId < allThreads)
        {
            auto j = binarySearchInsideRange(d_recalculateSequence, recalculateSequenceLength, threadId);
            auto k = threadId - d_recalculateSequence[j] + 1;
            printf("updating index: %u\n", d_startIndicies[j] + k * 255);
            d_startMask[d_startIndicies[j] + k * 255] = 1;
        }
    }

    // Helpers

    void compressCalculateScannedStartMask(uint32_t *d_startMask, uint32_t *d_scannedStartMask, size_t size)
    {
        thrust::inclusive_scan(thrust::device, d_startMask, d_startMask + size, d_scannedStartMask);
    }

    void compressRecalculateSequencePrescan(uint32_t *d_recalculateSequence, uint32_t size)
    {
        thrust::exclusive_scan(thrust::device, d_recalculateSequence, d_recalculateSequence + size, d_recalculateSequence);
    }

    __device__ size_t binarySearchInsideRange(uint32_t *d_arr, size_t size, uint32_t value)
    {
        size_t left = 0;
        size_t right = size - 1;

        while (left <= right)
        {
            size_t m = (left + right) / 2;
            if (d_arr[m] <= value)
            {
                if (m == size - 1 || d_arr[m + 1] >= value)
                {
                    return m;
                }
            }
            else if (d_arr[m] < value)
            {
                left = m + 1;
            }
            else if (d_arr[m] > value)
            {
                right = m - 1;
            }
        }

        return size;
    }

} // RunLength