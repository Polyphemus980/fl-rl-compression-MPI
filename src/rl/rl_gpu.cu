#include "hip/hip_runtime.h"
#include <thrust/execution_policy.h>
#include <thrust/scan.h>

#include "rl_gpu.cuh"
#include "../utils.cuh"

namespace RunLength
{
    // Main functions

    RLCompressed gpuCompress(uint8_t *data, size_t size)
    {
        if (size == 0)
        {
            return RLCompressed{
                .outputValues = nullptr,
                .outputCounts = nullptr,
                .count = 0};
        }

        // Copy input data to GPU
        uint8_t *d_data;
        CHECK_CUDA(hipMalloc(&d_data, sizeof(uint8_t) * size));
        CHECK_CUDA(hipMemcpy(d_data, data, sizeof(uint8_t) * size, hipMemcpyHostToDevice));

        // Prepare GPU arrays
        uint32_t *d_startMask;
        CHECK_CUDA(hipMalloc(&d_startMask, sizeof(uint32_t) * size));
        CHECK_CUDA(hipMemset(d_startMask, 0, sizeof(uint32_t) * size));
        uint32_t *d_scannedStartMask;
        CHECK_CUDA(hipMalloc(&d_scannedStartMask, sizeof(uint32_t) * size));
        uint32_t *d_startIndices;
        CHECK_CUDA(hipMalloc(&d_startIndices, sizeof(uint32_t) * size));
        uint32_t *d_startIndicesLength;
        CHECK_CUDA(hipMalloc(&d_startIndicesLength, sizeof(uint32_t)));
        // We could do it only after we know how much exactly we need, but it doesn't really matter
        // as we will copy back exact amount back to cpu anyway.
        // This way error handling is easier as all allocations are done at the beggining of the function.
        uint8_t *d_outputValues;
        CHECK_CUDA(hipMalloc(&d_outputValues, sizeof(uint8_t) * size));
        uint8_t *d_outputCounts;
        CHECK_CUDA(hipMalloc(&d_outputCounts, sizeof(uint8_t) * size));

        // Calculate start mask
        const uint32_t calculateStartMaskThreadsCount = 1024;
        const uint32_t calculateStartMaskBlocksCount = ceil(size * 1.0 / calculateStartMaskThreadsCount);
        compressCalculateStartMask<<<calculateStartMaskBlocksCount, calculateStartMaskThreadsCount>>>(d_data, size, d_startMask);
        CHECK_CUDA(hipDeviceSynchronize());
        CHECK_CUDA(hipGetLastError());

        // Calculate scanned start mask
        compressCalculateScannedStartMask(d_startMask, d_scannedStartMask, size);

        // Calculate start indicies
        const uint32_t calculateStartIndiciesThreadsCount = 1024;
        const uint32_t calculateStartIndiciesBlocksCount = ceil(size * 1.0 / calculateStartIndiciesThreadsCount);
        compressCalculateStartIndicies<<<calculateStartIndiciesBlocksCount, calculateStartIndiciesThreadsCount>>>(d_scannedStartMask, size, d_startIndices, d_startIndicesLength);
        CHECK_CUDA(hipDeviceSynchronize());
        CHECK_CUDA(hipGetLastError());

        // First copy to CPU size of final output to know how much bytes to copy (and allocate)
        // and to know how big kernel should be
        uint32_t outputSize = 0;
        CHECK_CUDA(hipMemcpy(&outputSize, d_startIndicesLength, sizeof(uint32_t), hipMemcpyDeviceToHost));

        // TODO: check if we need to recalculate some sequence due to size > 255

        // Calculate final output
        const uint32_t calculateOutputThreadsCount = 1024;
        const uint32_t calculateOutputBlocksCount = ceil(outputSize * 1.0 / calculateOutputThreadsCount);
        compressCalculateOutput<<<calculateOutputBlocksCount, calculateOutputThreadsCount>>>(d_data, size, d_startIndices, d_startIndicesLength, d_outputValues, d_outputCounts);

        // Wait for GPU to finish calculations
        CHECK_CUDA(hipDeviceSynchronize());

        // Allocate needed cpu arrays
        uint8_t *outputValues = reinterpret_cast<uint8_t *>(malloc(sizeof(uint8_t) * outputSize));
        if (outputValues == nullptr)
        {
            throw std::runtime_error("Cannot allocate memory");
        }
        uint8_t *outputCounts = reinterpret_cast<uint8_t *>(malloc(sizeof(uint8_t) * outputSize));
        if (outputCounts == nullptr)
        {
            throw std::runtime_error("Cannot allocate memory");
        }

        // Copy results to CPU
        CHECK_CUDA(hipMemcpy(outputValues, d_outputValues, sizeof(uint8_t) * outputSize, hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(outputCounts, d_outputCounts, sizeof(uint8_t) * outputSize, hipMemcpyDeviceToHost));

        // Deallocate GPU arrays
        hipFree(d_data);
        hipFree(d_startMask);
        hipFree(d_scannedStartMask);
        hipFree(d_startIndices);
        hipFree(d_startIndicesLength);
        hipFree(d_outputValues);
        hipFree(d_outputCounts);

        return RLCompressed{
            .outputValues = outputValues,
            .outputCounts = outputCounts,
            .count = outputSize,
        };
    }

    RLDecompressed gpuDecompress(uint8_t *values, uint8_t *counts, size_t size)
    {
        // TODO:
    }

    // Kernels

    __global__ void compressCalculateStartMask(uint8_t *d_data, size_t size, uint32_t *d_startMask)
    {
        auto threadId = blockDim.x * blockIdx.x + threadIdx.x;
        if (threadId == 0 || (threadId > 0 && threadId < size && d_data[threadId] != d_data[threadId - 1]))
        {
            d_startMask[threadId] = 1;
        }
    }

    __global__ void compressCalculateStartIndicies(uint32_t *d_scannedStartMask, size_t size, uint32_t *d_startIndicies, uint32_t *d_startIndiciesLength)
    {
        __shared__ uint32_t s_maxLength[1];
        auto threadId = blockDim.x * blockIdx.x + threadIdx.x;
        auto localThreadId = threadIdx.x;

        // Initialize shared memory
        if (localThreadId == 0)
        {
            s_maxLength[0] = 0;
        }
        __syncthreads();

        if (threadId == 0)
        {
            d_startIndicies[0] = 0;
        }
        else if (threadId < size && d_scannedStartMask[threadId] != d_scannedStartMask[threadId - 1])
        {
            auto id = d_scannedStartMask[threadId] - 1;
            d_startIndicies[id] = threadId;
            // + 1 because we want the length, not the index
            atomicMax(&s_maxLength[0], id + 1);
        }
        __syncthreads();

        // Save currently biggest changed index in global variable
        if (localThreadId == 0)
        {
            atomicMax(d_startIndiciesLength, s_maxLength[0]);
        }
    }

    __global__ void compressCalculateOutput(uint8_t *d_data, size_t size, uint32_t *d_startIndicies, uint32_t *d_startIndiciesLength, uint8_t *d_outputValues, uint8_t *d_outputCounts)
    {
        __shared__ uint32_t s_length[1];
        auto threadId = blockDim.x * blockIdx.x + threadIdx.x;
        auto localThreadId = threadIdx.x;

        // Initialize shared memory
        if (localThreadId == 0)
        {
            s_length[0] = d_startIndiciesLength[0];
        }
        __syncthreads();

        if (threadId < s_length[0])
        {
            d_outputValues[threadId] = d_data[d_startIndicies[threadId]];
        }

        if (threadId < s_length[0] - 1)
        {
            d_outputCounts[threadId] = d_startIndicies[threadId + 1] - d_startIndicies[threadId];
        }
        else if (threadId == s_length[0] - 1)
        {
            d_outputCounts[threadId] = (uint8_t)((uint32_t)size - d_startIndicies[threadId]);
        }
    }

    // Helpers

    void compressCalculateScannedStartMask(uint32_t *d_startMask, uint32_t *d_scannedStartMask, size_t size)
    {
        thrust::inclusive_scan(thrust::device, d_startMask, d_startMask + size, d_scannedStartMask);
    }

} // RunLength