#include "hip/hip_runtime.h"
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/scan.h>
#include <stdexcept>
#include <mpi.h>
#include "nccl.h"

#include "fl_gpu.cuh"
#include "fl_common.cuh"
#include "../utils.cuh"
#include "../timers/cpu_timer.cuh"
#include "../timers/gpu_timer.cuh"

namespace FixedLength
{
    FLCompressed DeviceToHost(const FLCompressedDevice &deviceData)
    {
        // Allocate host memory for the bits array
        uint8_t *h_outputBits = nullptr;
        if (deviceData.bitsSize > 0 && deviceData.d_outputBits != nullptr)
        {
            h_outputBits = new uint8_t[deviceData.bitsSize];
            // Copy from device to host
            hipMemcpy(h_outputBits, deviceData.d_outputBits, deviceData.bitsSize, hipMemcpyDeviceToHost);
        }

        // Allocate host memory for the values array
        uint8_t *h_outputValues = nullptr;
        if (deviceData.valuesSize > 0 && deviceData.d_outputValues != nullptr)
        {
            h_outputValues = new uint8_t[deviceData.valuesSize];
            // Copy from device to host
            hipMemcpy(h_outputValues, deviceData.d_outputValues, deviceData.valuesSize, hipMemcpyDeviceToHost);
        }

        // Create and return the host struct with copied data
        return FLCompressed(h_outputBits, deviceData.bitsSize, h_outputValues, deviceData.valuesSize, deviceData.inputSize);
    }

    FLCompressed gpuMPICompress(uint8_t *data, size_t size, MpiData mpiData)
    {
        Timers::CpuTimer cpuTimer;

        int rank = mpiData.rank;
        int nodesCount = mpiData.nodesCount;
        FLCompressed compressedData = gpuCompress(data, size);

        if (rank == 0)
        {
            FLCompressed *compressedWholeData = new FLCompressed[nodesCount];
            compressedWholeData[rank] = compressedData;
            cpuTimer.start();
            for (int i = 1; i < nodesCount; i++)
            {
                compressedWholeData[i] = FixedLength::FLCompressed::ReceiveFLCompressed(i, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
            }
            cpuTimer.end();
            cpuTimer.printResult("Receive compressed data from all nodes");
            MPI_Finalize();
            return FixedLength::FLCompressed::MergeFLCompressed(compressedWholeData, nodesCount);
        }
        else
        {
            cpuTimer.start();
            FixedLength::FLCompressed::SendFLCompressed(compressedData, 0, 0, MPI_COMM_WORLD);
            cpuTimer.end();
            cpuTimer.printResult("Send compressed data to node 0");
            MPI_Finalize();
            exit(0);
        }
    }

    FLCompressed gpuNCCLCompress(uint8_t *data, size_t size, MpiNcclData mpiNcclData)
    {
        Timers::CpuTimer cpuTimer;

        // Get the rank and size from the provided MpiNcclData
        int rank = mpiNcclData.rank;
        int nodesCount = mpiNcclData.nodesCount;
        ncclComm_t comm = mpiNcclData.comm;

        // Compress data on this GPU
        FLCompressedDevice compressedData = gpuCompressDevice(data, size);

        // We need to share metadata across all processes first
        // Create arrays to store metadata from all ranks
        size_t *all_bitsSizes = new size_t[nodesCount];
        size_t *all_valuesSizes = new size_t[nodesCount];
        size_t *all_inputSizes = new size_t[nodesCount];

        // Gather metadata using MPI since NCCL doesn't handle variable-sized data well
        MPI_Allgather(&compressedData.bitsSize, 1, MPI_UNSIGNED_LONG,
                    all_bitsSizes, 1, MPI_UNSIGNED_LONG, MPI_COMM_WORLD);
        MPI_Allgather(&compressedData.valuesSize, 1, MPI_UNSIGNED_LONG,
                    all_valuesSizes, 1, MPI_UNSIGNED_LONG, MPI_COMM_WORLD);
        MPI_Allgather(&compressedData.inputSize, 1, MPI_UNSIGNED_LONG,
                    all_inputSizes, 1, MPI_UNSIGNED_LONG, MPI_COMM_WORLD);

        // Calculate total sizes
        size_t total_bitsSize = 0;
        size_t total_valuesSize = 0;
        size_t total_inputSize = 0;

        for (int i = 0; i < nodesCount; i++)
        {
            total_bitsSize += all_bitsSizes[i];
            total_valuesSize += all_valuesSizes[i];
            total_inputSize += all_inputSizes[i];
        }

        // Find the maximum sizes for each buffer (for fixed-size communication)
        size_t max_bitsSize = 0;
        size_t max_valuesSize = 0;

        for (int i = 0; i < nodesCount; i++)
        {
            max_bitsSize = std::max(max_bitsSize, all_bitsSizes[i]);
            max_valuesSize = std::max(max_valuesSize, all_valuesSizes[i]);
        }

        // Allocate temporary buffers of the maximum size for gathering
        uint8_t *d_temp_bits = nullptr;
        uint8_t *d_temp_values = nullptr;

        if (max_bitsSize > 0)
        {
            CHECK_CUDA(hipMalloc(&d_temp_bits, max_bitsSize * nodesCount));
            CHECK_CUDA(hipMemset(d_temp_bits, 0, max_bitsSize * nodesCount));
        }

        if (max_valuesSize > 0)
        {
            CHECK_CUDA(hipMalloc(&d_temp_values, max_valuesSize * nodesCount));
            CHECK_CUDA(hipMemset(d_temp_values, 0, max_valuesSize * nodesCount));
        }

        // Gather all data using NCCL
        cpuTimer.start();

        // AllGather for bits
        if (max_bitsSize > 0)
        {
            // Ensure all processes use the same size buffer, padded with zeros if necessary
            uint8_t *d_padded_bits = nullptr;
            CHECK_CUDA(hipMalloc(&d_padded_bits, max_bitsSize));
            CHECK_CUDA(hipMemset(d_padded_bits, 0, max_bitsSize));

            if (compressedData.bitsSize > 0)
            {
                CHECK_CUDA(hipMemcpy(d_padded_bits, compressedData.d_outputBits,
                                    compressedData.bitsSize, hipMemcpyDeviceToDevice));
            }

            ncclAllGather(d_padded_bits, d_temp_bits, max_bitsSize,
                        ncclUint8, comm, nullptr);

            CHECK_CUDA(hipFree(d_padded_bits));
        }

        // AllGather for values
        if (max_valuesSize > 0)
        {
            // Ensure all processes use the same size buffer, padded with zeros if necessary
            uint8_t *d_padded_values = nullptr;
            CHECK_CUDA(hipMalloc(&d_padded_values, max_valuesSize));
            CHECK_CUDA(hipMemset(d_padded_values, 0, max_valuesSize));

            if (compressedData.valuesSize > 0)
            {
                CHECK_CUDA(hipMemcpy(d_padded_values, compressedData.d_outputValues,
                                    compressedData.valuesSize, hipMemcpyDeviceToDevice));
            }

            ncclAllGather(d_padded_values, d_temp_values, max_valuesSize,
                        ncclUint8, comm, nullptr);

            CHECK_CUDA(hipFree(d_padded_values));
        }

        // Ensure all NCCL operations are complete
        CHECK_CUDA(hipDeviceSynchronize());

        // Only rank 0 will process the merged data
        FLCompressed result;
        
        if (rank == 0)
        {
            // Allocate memory for the merged data (without padding)
            uint8_t *d_mergedBits = nullptr;
            uint8_t *d_mergedValues = nullptr;

            if (total_bitsSize > 0)
            {
                CHECK_CUDA(hipMalloc(&d_mergedBits, total_bitsSize));
            }

            if (total_valuesSize > 0)
            {
                CHECK_CUDA(hipMalloc(&d_mergedValues, total_valuesSize));
            }

            // Copy from the temporary padded buffers to the final unpadded merged buffers
            size_t bits_offset = 0;
            size_t values_offset = 0;

            for (int i = 0; i < nodesCount; i++)
            {
                if (all_bitsSizes[i] > 0)
                {
                    CHECK_CUDA(hipMemcpy(d_mergedBits + bits_offset,
                                        d_temp_bits + (i * max_bitsSize),
                                        all_bitsSizes[i], hipMemcpyDeviceToDevice));
                    bits_offset += all_bitsSizes[i];
                }

                if (all_valuesSizes[i] > 0)
                {
                    CHECK_CUDA(hipMemcpy(d_mergedValues + values_offset,
                                        d_temp_values + (i * max_valuesSize),
                                        all_valuesSizes[i], hipMemcpyDeviceToDevice));
                    values_offset += all_valuesSizes[i];
                }
            }

            cpuTimer.end();
            cpuTimer.printResult("NCCL gather compressed data from all nodes");

            // Create the merged compressed data
            auto merged = FLCompressedDevice(d_mergedBits, total_bitsSize, d_mergedValues, total_valuesSize, total_inputSize);
            result = DeviceToHost(merged);
        }

        // Clean up
        if (d_temp_bits)
            CHECK_CUDA(hipFree(d_temp_bits));
        if (d_temp_values)
            CHECK_CUDA(hipFree(d_temp_values));

        // Free the original compressed data on device
        if (compressedData.d_outputBits)
        {
            CHECK_CUDA(hipFree(compressedData.d_outputBits));
        }
        if (compressedData.d_outputValues)
        {
            CHECK_CUDA(hipFree(compressedData.d_outputValues));
        }

        delete[] all_bitsSizes;
        delete[] all_valuesSizes;
        delete[] all_inputSizes;

        // Finalize MPI for all processes
        MPI_Finalize();
        
        // Non-root processes exit after cleanup
        if (rank != 0)
        {
            exit(0);
        }

        return result;
    }

    // Main functions
    FLCompressed gpuCompress(uint8_t *data, size_t size)
    {
        if (size == 0)
        {
            return FLCompressed();
        }

        std::exception error;
        bool isError = false;

        Timers::CpuTimer cpuTimer;
        Timers::GpuTimer gpuTimer;

        size_t bitsSize = ceil(size * 1.0 / FRAME_LENGTH);
        size_t valuesSize = 0;

        // GPU arrays
        uint8_t *d_data = nullptr;
        uint8_t *d_outputBits = nullptr;
        uint64_t *d_frameStartIndiciesBits = nullptr;
        uint8_t *d_outputValues = nullptr;

        // CPU arrays
        uint8_t *outputBits = nullptr;
        uint8_t *outputValues = nullptr;

        try
        {
            gpuTimer.start();

            // Allocate arrays on GPU
            CHECK_CUDA(hipMalloc(&d_data, sizeof(uint8_t) * size));
            CHECK_CUDA(hipMalloc(&d_outputBits, sizeof(uint8_t) * bitsSize));
            CHECK_CUDA(hipMalloc(&d_frameStartIndiciesBits, sizeof(uint64_t) * bitsSize));

            gpuTimer.end();
            gpuTimer.printResult("Allocate arrays on GPU");

            gpuTimer.start();

            // Copy input to GPU
            CHECK_CUDA(hipMemcpy(d_data, data, sizeof(uint8_t) * size, hipMemcpyHostToDevice));

            gpuTimer.end();
            gpuTimer.printResult("Copy input data to GPU");

            gpuTimer.start();

            // Calculate outputBits
            constexpr size_t outputBitsThreadsPerBlock = BLOCK_SIZE;
            const size_t outputBitsBlocksCount = ceil(size * 1.0 / outputBitsThreadsPerBlock);
            compressCalculateOutputBits<<<outputBitsBlocksCount, outputBitsThreadsPerBlock>>>(d_data, size, d_outputBits, bitsSize);
            CHECK_CUDA(hipDeviceSynchronize());
            CHECK_CUDA(hipGetLastError());

            // Calculate frameStartIndiciesBits
            constexpr size_t frameStartIndiciesThreadsPerBlock = BLOCK_SIZE;
            const size_t frameStartIndiciesBlocksCount = ceil(bitsSize * 1.0 / frameStartIndiciesThreadsPerBlock);
            compressInitializeFrameStartIndiciesBits<<<frameStartIndiciesBlocksCount, frameStartIndiciesThreadsPerBlock>>>(d_frameStartIndiciesBits, d_outputBits, bitsSize);
            CHECK_CUDA(hipDeviceSynchronize());
            CHECK_CUDA(hipGetLastError());
            compressCalculateFrameStartIndiciesBits(d_frameStartIndiciesBits, bitsSize);

            // Calculate length of outputValues array
            uint8_t outputBitsLast = 0;
            CHECK_CUDA(hipMemcpy(&outputBitsLast, &d_outputBits[bitsSize - 1], sizeof(uint8_t), hipMemcpyDeviceToHost));
            uint64_t frameStartIndiciesBitsLast = 0;
            CHECK_CUDA(hipMemcpy(&frameStartIndiciesBitsLast, &d_frameStartIndiciesBits[bitsSize - 1], sizeof(uint64_t), hipMemcpyDeviceToHost));
            uint64_t lastFrameElementCount = size % FRAME_LENGTH == 0 ? FRAME_LENGTH : (size - (size / FRAME_LENGTH) * FRAME_LENGTH);
            valuesSize = ceil((frameStartIndiciesBitsLast + lastFrameElementCount * outputBitsLast) * 1.0 / 8);

            // Allocate gpu array for `outputValues`
            CHECK_CUDA(hipMalloc(&d_outputValues, sizeof(uint8_t) * valuesSize));
            CHECK_CUDA(hipMemset(d_outputValues, 0, sizeof(uint8_t) * valuesSize));

            constexpr size_t outputValuesThreadsPerBlock = BLOCK_SIZE;
            const size_t outputValuesBlocksCount = ceil(size * 1.0 / outputValuesThreadsPerBlock);
            compressCalculateOutput<<<outputValuesBlocksCount, outputValuesThreadsPerBlock>>>(d_data, size, d_outputBits, bitsSize, d_frameStartIndiciesBits, d_outputValues, valuesSize);
            CHECK_CUDA(hipDeviceSynchronize());
            CHECK_CUDA(hipGetLastError());

            gpuTimer.end();
            gpuTimer.printResult("Compression");

            cpuTimer.start();

            // Allocate arrays on CPU
            outputBits = reinterpret_cast<uint8_t *>(malloc(sizeof(uint8_t) * bitsSize));
            if (outputBits == nullptr)
            {
                throw std::runtime_error("Cannot allocate memory");
            }
            outputValues = reinterpret_cast<uint8_t *>(malloc(sizeof(uint8_t) * valuesSize));
            if (outputValues == nullptr)
            {
                throw std::runtime_error("Cannot allocate memory");
            }

            cpuTimer.end();
            cpuTimer.printResult("Allocate arrays on CPU");

            gpuTimer.start();

            // Copy results to CPU
            CHECK_CUDA(hipMemcpy(outputBits, d_outputBits, sizeof(uint8_t) * bitsSize, hipMemcpyDeviceToHost));
            CHECK_CUDA(hipMemcpy(outputValues, d_outputValues, sizeof(uint8_t) * valuesSize, hipMemcpyDeviceToHost));

            gpuTimer.end();
            gpuTimer.printResult("Copy results to CPU");
        }
        catch (const std::exception &e)
        {
            error = e;
            isError = true;
        }

        gpuTimer.start();

        // Deallocate gpu arrays
        hipFree(d_data);
        hipFree(d_outputBits);
        hipFree(d_frameStartIndiciesBits);
        hipFree(d_outputValues);

        gpuTimer.end();
        gpuTimer.printResult("Deallocate ararys on GPU");

        if (isError)
        {
            throw error;
        }

        return FLCompressed(outputBits, bitsSize, outputValues, valuesSize, size);
    }

    FLCompressedDevice gpuCompressDevice(uint8_t *data, size_t size)
    {
        if (size == 0)
        {
            return FLCompressedDevice();
        }

        std::exception error;
        bool isError = false;

        Timers::CpuTimer cpuTimer;
        Timers::GpuTimer gpuTimer;

        size_t bitsSize = ceil(size * 1.0 / FRAME_LENGTH);
        size_t valuesSize = 0;

        // GPU arrays
        uint8_t *d_data = nullptr;
        uint8_t *d_outputBits = nullptr;
        uint64_t *d_frameStartIndiciesBits = nullptr;
        uint8_t *d_outputValues = nullptr;

        try
        {
            gpuTimer.start();

            // Allocate arrays on GPU
            CHECK_CUDA(hipMalloc(&d_data, sizeof(uint8_t) * size));
            CHECK_CUDA(hipMalloc(&d_outputBits, sizeof(uint8_t) * bitsSize));
            CHECK_CUDA(hipMalloc(&d_frameStartIndiciesBits, sizeof(uint64_t) * bitsSize));

            gpuTimer.end();
            gpuTimer.printResult("Allocate arrays on GPU");

            gpuTimer.start();

            // Copy input to GPU
            CHECK_CUDA(hipMemcpy(d_data, data, sizeof(uint8_t) * size, hipMemcpyHostToDevice));

            gpuTimer.end();
            gpuTimer.printResult("Copy input data to GPU");

            gpuTimer.start();

            // Calculate outputBits
            constexpr size_t outputBitsThreadsPerBlock = BLOCK_SIZE;
            const size_t outputBitsBlocksCount = ceil(size * 1.0 / outputBitsThreadsPerBlock);
            compressCalculateOutputBits<<<outputBitsBlocksCount, outputBitsThreadsPerBlock>>>(d_data, size, d_outputBits, bitsSize);
            CHECK_CUDA(hipDeviceSynchronize());
            CHECK_CUDA(hipGetLastError());

            // Calculate frameStartIndiciesBits
            constexpr size_t frameStartIndiciesThreadsPerBlock = BLOCK_SIZE;
            const size_t frameStartIndiciesBlocksCount = ceil(bitsSize * 1.0 / frameStartIndiciesThreadsPerBlock);
            compressInitializeFrameStartIndiciesBits<<<frameStartIndiciesBlocksCount, frameStartIndiciesThreadsPerBlock>>>(d_frameStartIndiciesBits, d_outputBits, bitsSize);
            CHECK_CUDA(hipDeviceSynchronize());
            CHECK_CUDA(hipGetLastError());
            compressCalculateFrameStartIndiciesBits(d_frameStartIndiciesBits, bitsSize);

            // Calculate length of outputValues array
            uint8_t outputBitsLast = 0;
            CHECK_CUDA(hipMemcpy(&outputBitsLast, &d_outputBits[bitsSize - 1], sizeof(uint8_t), hipMemcpyDeviceToHost));
            uint64_t frameStartIndiciesBitsLast = 0;
            CHECK_CUDA(hipMemcpy(&frameStartIndiciesBitsLast, &d_frameStartIndiciesBits[bitsSize - 1], sizeof(uint64_t), hipMemcpyDeviceToHost));
            uint64_t lastFrameElementCount = size % FRAME_LENGTH == 0 ? FRAME_LENGTH : (size - (size / FRAME_LENGTH) * FRAME_LENGTH);
            valuesSize = ceil((frameStartIndiciesBitsLast + lastFrameElementCount * outputBitsLast) * 1.0 / 8);

            // Allocate gpu array for `outputValues`
            CHECK_CUDA(hipMalloc(&d_outputValues, sizeof(uint8_t) * valuesSize));
            CHECK_CUDA(hipMemset(d_outputValues, 0, sizeof(uint8_t) * valuesSize));

            constexpr size_t outputValuesThreadsPerBlock = BLOCK_SIZE;
            const size_t outputValuesBlocksCount = ceil(size * 1.0 / outputValuesThreadsPerBlock);
            compressCalculateOutput<<<outputValuesBlocksCount, outputValuesThreadsPerBlock>>>(d_data, size, d_outputBits, bitsSize, d_frameStartIndiciesBits, d_outputValues, valuesSize);
            CHECK_CUDA(hipDeviceSynchronize());
            CHECK_CUDA(hipGetLastError());

            gpuTimer.end();
            gpuTimer.printResult("Compression");

            // Free temporary device memory that's no longer needed
            hipFree(d_data);
            hipFree(d_frameStartIndiciesBits);

            // Return the FLCompressedDevice structure with device pointers
            return FLCompressedDevice(d_outputBits, bitsSize, d_outputValues, valuesSize, size);
        }
        catch (const std::exception &e)
        {
            error = e;
            isError = true;
            std::cout << e.what() << std::endl;
        }

        // Handle error case - cleanup and throw
        if (d_data != nullptr)
            hipFree(d_data);
        if (d_outputBits != nullptr)
            hipFree(d_outputBits);
        if (d_frameStartIndiciesBits != nullptr)
            hipFree(d_frameStartIndiciesBits);
        if (d_outputValues != nullptr)
            hipFree(d_outputValues);

        if (isError)
        {
            throw error;
        }

        return FLCompressedDevice();
    }

    FLDecompressed gpuDecompress(size_t outputSize, uint8_t *bits, size_t bitsSize, uint8_t *values, size_t valuesSize)
    {
        if (valuesSize == 0 || bitsSize == 0 || outputSize == 0)
        {
            return FLDecompressed();
        }

        Timers::CpuTimer cpuTimer;
        Timers::GpuTimer gpuTimer;

        std::exception error;
        bool isError = false;

        // CPU arrays
        uint8_t *data;

        // GPU arrays
        uint8_t *d_bits;
        uint8_t *d_values;
        uint64_t *d_frameStartIndiciesBits;
        uint8_t *d_data;

        try
        {
            cpuTimer.start();

            // Allocate array on CPU
            data = reinterpret_cast<uint8_t *>(malloc(sizeof(uint8_t) * outputSize));
            if (data == nullptr)
            {
                throw std::runtime_error("Cannot allocate memory");
            }

            cpuTimer.end();
            cpuTimer.printResult("Allocate arrays on CPU");

            gpuTimer.start();

            // Allocate arrays on GPU
            CHECK_CUDA(hipMalloc(&d_bits, sizeof(uint8_t) * bitsSize));
            CHECK_CUDA(hipMalloc(&d_values, sizeof(uint8_t) * valuesSize));
            CHECK_CUDA(hipMalloc(&d_frameStartIndiciesBits, sizeof(uint64_t) * bitsSize));
            CHECK_CUDA(hipMalloc(&d_data, sizeof(uint8_t) * outputSize));

            gpuTimer.end();
            gpuTimer.printResult("Allocate arrays on GPU");

            gpuTimer.start();

            // Copy input to GPU
            CHECK_CUDA(hipMemcpy(d_bits, bits, sizeof(uint8_t) * bitsSize, hipMemcpyHostToDevice));
            CHECK_CUDA(hipMemcpy(d_values, values, sizeof(uint8_t) * valuesSize, hipMemcpyHostToDevice));

            gpuTimer.end();
            gpuTimer.printResult("Copy input to GPU");

            gpuTimer.start();

            // Calculate frameStartIndiciesBits
            constexpr size_t frameStartIndiciesThreadsPerBlock = BLOCK_SIZE;
            const size_t frameStartIndiciesBlocksCount = ceil(bitsSize * 1.0 / frameStartIndiciesThreadsPerBlock);
            compressInitializeFrameStartIndiciesBits<<<frameStartIndiciesBlocksCount, frameStartIndiciesThreadsPerBlock>>>(d_frameStartIndiciesBits, d_bits, bitsSize);
            CHECK_CUDA(hipDeviceSynchronize());
            CHECK_CUDA(hipGetLastError());
            compressCalculateFrameStartIndiciesBits(d_frameStartIndiciesBits, bitsSize);

            // Calculate output
            constexpr size_t outputThreadsPerBlock = BLOCK_SIZE;
            const size_t outputBlocksCount = ceil(outputSize * 1.0 / outputThreadsPerBlock);
            decompressCalculateOutput<<<outputBlocksCount, outputThreadsPerBlock>>>(d_data, outputSize, d_bits, bitsSize, d_values, valuesSize, d_frameStartIndiciesBits);
            CHECK_CUDA(hipDeviceSynchronize());
            CHECK_CUDA(hipGetLastError());

            gpuTimer.end();
            gpuTimer.printResult("Decompression");

            gpuTimer.start();

            // Copy result to CPU
            CHECK_CUDA(hipMemcpy(data, d_data, sizeof(uint8_t) * outputSize, hipMemcpyDeviceToHost));

            gpuTimer.end();
            gpuTimer.printResult("Copy results to CPU");
        }
        catch (const std::exception &e)
        {
            error = e;
            isError = true;
        }

        gpuTimer.start();

        // Deallocate GPU arrays
        hipFree(d_bits);
        hipFree(d_values);
        hipFree(d_frameStartIndiciesBits);
        hipFree(d_data);

        gpuTimer.end();
        gpuTimer.printResult("Deallocate arrays on GPU");

        if (isError)
        {
            throw error;
        }

        return FLDecompressed(data, outputSize);
    }

    // Kernels
    __global__ void compressCalculateOutputBits(uint8_t *d_data, size_t size, uint8_t *d_outputBits, size_t bitsSize)
    {
        auto threadId = blockDim.x * blockIdx.x + threadIdx.x;
        auto localThreadId = threadIdx.x;

        // Don't follow if threadId is outside of data scope
        if (threadId >= size)
        {
            return;
        }

        constexpr size_t FRAMES_PER_BLOCK = BLOCK_SIZE / FRAME_LENGTH;

        auto frameId = threadId / FRAME_LENGTH;
        auto localFrameId = frameId - blockIdx.x * FRAMES_PER_BLOCK;

        __shared__ uint8_t s_outputBits[FRAMES_PER_BLOCK];

        // Initialize shared memory
        // We always need at least 1
        if (localThreadId < FRAMES_PER_BLOCK)
        {
            s_outputBits[localThreadId] = 1;
        }
        __syncthreads();

        // Calculate number of required bits
        uint8_t requiredBits = 8 - countLeadingZeroes(d_data[threadId]);
        atomicMaxUint8t(&s_outputBits[localFrameId], requiredBits);
        __syncthreads();

        // Push results back to global memoryd_frameStartIndiciesBits
        auto globalId = blockIdx.x * FRAMES_PER_BLOCK + localThreadId;
        if (localThreadId < FRAMES_PER_BLOCK && globalId < bitsSize)
        {
            atomicMaxUint8t(&d_outputBits[globalId], s_outputBits[localThreadId]);
        }
    }

    __global__ void compressInitializeFrameStartIndiciesBits(uint64_t *d_frameStartIndiciesBits, uint8_t *d_outputBits, size_t bitsSize)
    {
        auto threadId = blockDim.x * blockIdx.x + threadIdx.x;

        // Don't follow if threadId is outside of data scope
        if (threadId >= bitsSize)
        {
            return;
        }

        d_frameStartIndiciesBits[threadId] = d_outputBits[threadId] * FRAME_LENGTH;
    }

    __global__ void compressCalculateOutput(uint8_t *d_data, size_t size, uint8_t *d_outputBits, size_t bitsSize, uint64_t *d_frameStartIndiciesBits, uint8_t *d_outputValues, size_t valuesSize)
    {
        auto threadId = blockDim.x * blockIdx.x + threadIdx.x;

        // Don't follow if threadId is outside of data scope
        if (threadId >= size)
        {
            return;
        }

        // Encode data
        uint64_t frameId = threadId / FRAME_LENGTH;
        uint64_t frameElementId = threadId % FRAME_LENGTH;
        uint8_t requiredBits = d_outputBits[frameId];
        uint64_t bitsOffset = d_frameStartIndiciesBits[frameId] + frameElementId * (uint64_t)requiredBits;
        size_t outputId = bitsOffset / 8;
        uint8_t outputOffset = bitsOffset % 8;
        uint8_t value = d_data[threadId];
        uint8_t encodedValue = (value << outputOffset);
        atomicOrUint8t(&d_outputValues[outputId], encodedValue);
        // If it overflows encode the overflowed part on next byte
        if (outputOffset + requiredBits > 8)
        {
            uint8_t overflowValue = (value >> (8 - outputOffset));
            atomicOrUint8t(&d_outputValues[outputId + 1], overflowValue);
        }
    }

    __global__ void decompressCalculateOutput(uint8_t *d_data, size_t size, uint8_t *d_bits, size_t bitsSize, uint8_t *d_values, size_t valuesSize, uint64_t *d_frameStartIndiciesBits)
    {
        auto threadId = blockDim.x * blockIdx.x + threadIdx.x;
        // Don't follow if threadId is outside of data scope
        if (threadId >= size)
        {
            return;
        }

        // Decode data
        uint64_t frameId = threadId / FRAME_LENGTH;
        uint64_t frameElementId = threadId % FRAME_LENGTH;
        uint8_t usedBits = d_bits[frameId];
        uint64_t bitsOffset = d_frameStartIndiciesBits[frameId] + frameElementId * usedBits;
        size_t inputId = bitsOffset / 8;
        uint8_t inputOffset = bitsOffset % 8;
        uint8_t mask = (1 << usedBits) - 1;
        uint8_t decodedValue = (d_values[inputId] >> inputOffset) & mask;
        // If it overflow decode the overflowed part of the next byte
        if (inputOffset + usedBits > 8)
        {
            uint8_t overflowBits = inputOffset + usedBits - 8;
            uint8_t overflowMask = (1 << overflowBits) - 1;
            uint8_t overflowValue = (d_values[inputId + 1] & overflowMask) << (usedBits - overflowBits);
            decodedValue |= overflowValue;
        }
        d_data[threadId] = decodedValue;
    }

    // Helpers
    __device__ uint8_t atomicMaxUint8t(uint8_t *address, uint8_t val)
    {
        unsigned int *base_address = (unsigned int *)((size_t)address & ~3);
        unsigned int selectors[] = {0x3214, 0x3240, 0x3410, 0x4210};
        unsigned int sel = selectors[(size_t)address & 3];
        unsigned int old, assumed, max_, new_;
        old = *base_address;
        do
        {
            assumed = old;
            max_ = max(val, (uint8_t)__byte_perm(old, 0, ((size_t)address & 3) | 0x4440));
            new_ = __byte_perm(old, max_, sel);

            if (new_ == old)
                break;

            old = atomicCAS(base_address, assumed, new_);

        } while (assumed != old);

        return old;
    }

    __device__ uint8_t atomicOrUint8t(uint8_t *address, uint8_t val)
    {
        unsigned int *base_address = (unsigned int *)((size_t)address & ~3);
        unsigned int selectors[] = {0x3214, 0x3240, 0x3410, 0x4210};
        unsigned int sel = selectors[(size_t)address & 3];
        unsigned int old, assumed, new_, current_val, updated_val;
        old = *base_address;
        do
        {
            assumed = old;
            current_val = (uint8_t)__byte_perm(old, 0, ((size_t)address & 3) | 0x4440);
            updated_val = current_val | val;
            new_ = __byte_perm(old, updated_val, sel);

            if (new_ == old)
                break;

            old = atomicCAS(base_address, assumed, new_);

        } while (assumed != old);

        return old;
    }

    void compressCalculateFrameStartIndiciesBits(uint64_t *d_frameStartIndiciesBits, size_t bitsSize)
    {
        thrust::exclusive_scan(thrust::device, d_frameStartIndiciesBits, d_frameStartIndiciesBits + bitsSize, d_frameStartIndiciesBits);
    }

}