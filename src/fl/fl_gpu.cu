#include "hip/hip_runtime.h"
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/scan.h>

#include "fl_gpu.cuh"
#include "../utils.cuh"

namespace FixedLength
{
    // Main functions
    FLCompressed gpuCompress(uint8_t *data, size_t size)
    {
        if (size == 0)
        {
            return FLCompressed{
                .outputBits = nullptr,
                .bitsSize = 0,
                .outputValues = nullptr,
                .valuesSize = 0,
                .inputSize = 0};
        }

        // Copy input to GPU
        uint8_t *d_data;
        CHECK_CUDA(hipMalloc(&d_data, sizeof(uint8_t) * size));
        CHECK_CUDA(hipMemcpy(d_data, data, sizeof(uint8_t) * size, hipMemcpyHostToDevice));

        // Allocate arrays on GPU
        size_t bitsSize = ceil(size * 1.0 / FRAME_LENGTH);
        uint8_t *d_outputBits;
        CHECK_CUDA(hipMalloc(&d_outputBits, sizeof(uint8_t) * bitsSize));
        uint64_t *d_frameStartIndiciesBits;
        CHECK_CUDA(hipMalloc(&d_frameStartIndiciesBits, sizeof(uint64_t) * bitsSize));

        // Calculate outputBits
        constexpr size_t outputBitsThreadsPerBlock = BLOCK_SIZE;
        const size_t outputBitsBlocksCount = ceil(size * 1.0 / outputBitsThreadsPerBlock);
        compressCalculateOutputBits<<<outputBitsBlocksCount, outputBitsThreadsPerBlock>>>(d_data, size, d_outputBits, bitsSize);
        CHECK_CUDA(hipDeviceSynchronize());
        CHECK_CUDA(hipGetLastError());

        // FIXME: remove me, only for testing
        {
            uint8_t *outputBitsCPU = reinterpret_cast<uint8_t *>(malloc(sizeof(uint8_t) * bitsSize));
            CHECK_CUDA(hipMemcpy(outputBitsCPU, d_outputBits, sizeof(uint8_t) * bitsSize, hipMemcpyDeviceToHost));
            printf("bits size: %lu\n", bitsSize);
            printf("output Bits: \n");
            for (size_t i = 0; i < bitsSize; i++)
            {
                printf("%hhu\n", outputBitsCPU[i]);
            }
        }

        // Calculate frameStartIndiciesBits
        constexpr size_t frameStartIndiciesThreadsPerBlock = BLOCK_SIZE;
        const size_t frameStartIndiciesBlocksCount = ceil(bitsSize * 1.0 / frameStartIndiciesThreadsPerBlock);
        compressInitializeFrameStartIndiciesBits<<<frameStartIndiciesBlocksCount, frameStartIndiciesThreadsPerBlock>>>(d_frameStartIndiciesBits, d_outputBits, bitsSize);
        CHECK_CUDA(hipDeviceSynchronize());
        CHECK_CUDA(hipGetLastError());
        compressCalculateFrameStartIndiciesBits(d_frameStartIndiciesBits, bitsSize);

        // FIXME: remove me, only for testing
        {
            uint64_t *frameStartIndiciesBitsCPU = reinterpret_cast<uint64_t *>(malloc(sizeof(uint64_t) * bitsSize));
            CHECK_CUDA(hipMemcpy(frameStartIndiciesBitsCPU, d_frameStartIndiciesBits, sizeof(uint64_t) * bitsSize, hipMemcpyDeviceToHost));
            printf("frameStartIndiciesBits: \n");
            for (size_t i = 0; i < bitsSize; i++)
            {
                printf("%lu\n", frameStartIndiciesBitsCPU[i]);
            }
        }

        // Calculate length of outputValues array
        uint8_t outputBitsLast = 0;
        CHECK_CUDA(hipMemcpy(&outputBitsLast, &d_outputBits[bitsSize - 1], sizeof(uint8_t), hipMemcpyDeviceToHost));
        uint64_t frameStartIndiciesBitsLast = 0;
        CHECK_CUDA(hipMemcpy(&frameStartIndiciesBitsLast, &d_frameStartIndiciesBits[bitsSize - 1], sizeof(uint64_t), hipMemcpyDeviceToHost));
        uint64_t lastFrameElementCount = size % FRAME_LENGTH == 0 ? FRAME_LENGTH : (size - (size / FRAME_LENGTH) * FRAME_LENGTH);
        size_t valuesSize = ceil((frameStartIndiciesBitsLast + lastFrameElementCount * outputBitsLast) * 1.0 / 8);

        // FIXME: remove me, only for testing
        {
            printf("values size: %lu\n", valuesSize);
        }

        // TODO: finish

        // Deallocate gpu arrays
        hipFree(d_data);
        hipFree(d_outputBits);
        hipFree(d_frameStartIndiciesBits);

        // TODO: fill it
        return FLCompressed{
            .outputBits = nullptr,
            .bitsSize = bitsSize,
            .outputValues = nullptr,
            .valuesSize = valuesSize,
            .inputSize = size};
    }

    // Kernels
    __global__ void compressCalculateOutputBits(uint8_t *d_data, size_t size, uint8_t *d_outputBits, size_t bitsSize)
    {
        auto threadId = blockDim.x * blockIdx.x + threadIdx.x;
        auto localThreadId = threadIdx.x;

        // Don't follow if threadId is outside of data scope
        if (threadId >= size)
        {
            return;
        }

        constexpr size_t FRAMES_PER_BLOCK = BLOCK_SIZE / FRAME_LENGTH;

        auto frameId = threadId / FRAME_LENGTH;
        auto localFrameId = frameId - blockIdx.x * FRAMES_PER_BLOCK;

        __shared__ uint8_t s_outputBits[FRAMES_PER_BLOCK];

        // Initialize shared memory
        // We always need at least 1
        if (localThreadId < FRAMES_PER_BLOCK)
        {
            s_outputBits[localThreadId] = 1;
        }
        __syncthreads();

        // Calculate number of required bits
        uint8_t requiredBits = 8 - countLeadingZeroes(d_data[threadId]);
        atomicMaxUint8t(&s_outputBits[localFrameId], requiredBits);
        __syncthreads();

        // Push results back to global memory
        auto globalId = blockIdx.x * FRAMES_PER_BLOCK + localThreadId;
        if (localThreadId < FRAMES_PER_BLOCK && globalId < bitsSize)
        {
            atomicMaxUint8t(&d_outputBits[globalId], s_outputBits[localThreadId]);
        }
    }

    __global__ void compressInitializeFrameStartIndiciesBits(uint64_t *d_frameStartIndiciesBits, uint8_t *d_outputBits, size_t bitsSize)
    {
        auto threadId = blockDim.x * blockIdx.x + threadIdx.x;

        // Don't follow if threadId is outside of data scope
        if (threadId >= bitsSize)
        {
            return;
        }

        d_frameStartIndiciesBits[threadId] = d_outputBits[threadId] * FRAME_LENGTH;
    }

    __global__ void compressCalculateOutput(uint8_t *d_data, size_t size, uint8_t *d_outputBits, size_t bitsSize, uint64_t *d_frameStartIndiciesBits, uint8_t *outputValues, size_t valuesSize)
    {
        auto threadId = blockDim.x * blockIdx.x + threadIdx.x;
        auto localThreadId = threadIdx.x;

        // Don't follow if threadId is outside of data scope
        if (threadId >= size)
        {
            return;
        }

        // This should be the same size as `outputValues`
        extern __shared__ uint8_t s_outputValues[];

        // Initialize shared memory
        size_t toInitPerThread = valuesSize / blockDim.x;
        size_t forLastThreadAdditional = valuesSize % blockDim.x;
        // TODO:
        __syncthreads();

        // Encode data
        uint64_t frameId = threadId / FRAME_LENGTH;
        uint64_t frameElementId = threadId % FRAME_LENGTH;
        uint8_t requiredBits = d_outputBits[frameId];
        uint64_t bitsOffset = frameId * FRAME_LENGTH * 8 + frameElementId * requiredBits;
        size_t outputId = bitsOffset / 8;
        uint8_t outputOffset = bitsOffset % 8;
        uint8_t value = d_data[threadId];
        uint8_t encodedValue = value << outputOffset;
        // TODO: Save value to shared memory
        // If it overflows encode the overflowed part on next byte
        if (outputOffset + requiredBits > 8)
        {
            uint8_t overflowValue = value >> (8 - outputOffset);
            // TODO: Save value to shared memory
        }
        __syncthreads();

        // Save result to global memory
        // TODO:
    }

    // Helpers
    __device__ uint8_t atomicMaxUint8t(uint8_t *address, uint8_t val)
    {
        unsigned int *base_address = (unsigned int *)((size_t)address & ~3);
        unsigned int selectors[] = {0x3214, 0x3240, 0x3410, 0x4210};
        unsigned int sel = selectors[(size_t)address & 3];
        unsigned int old, assumed, max_, new_;
        old = *base_address;
        do
        {
            assumed = old;
            max_ = max(val, (uint8_t)__byte_perm(old, 0, ((size_t)address & 3) | 0x4440));
            new_ = __byte_perm(old, max_, sel);

            if (new_ == old)
                break;

            old = atomicCAS(base_address, assumed, new_);

        } while (assumed != old);

        return old;
    }

    __device__ uint8_t atomicOrUint8t(uint8_t *address, uint8_t val)
    {
        unsigned int *base_address = (unsigned int *)((size_t)address & ~3);
        unsigned int byte_position = (size_t)address & 3;
        unsigned int selectors[] = {0x3210, 0x3204, 0x3404, 0x4204};
        unsigned int sel = selectors[byte_position];
        unsigned int old, assumed, new_;
        old = *base_address;
        do
        {
            assumed = old;
            uint8_t current_val = (uint8_t)__byte_perm(old, 0, byte_position | 0x4440);
            uint8_t updated_val = current_val | val;
            new_ = __byte_perm(old, updated_val, sel);

            if (new_ == old)
                break;

            old = atomicCAS(base_address, assumed, new_);

        } while (assumed != old);

        return (uint8_t)__byte_perm(old, 0, byte_position | 0x4440);
    }

    void compressCalculateFrameStartIndiciesBits(uint64_t *d_frameStartIndiciesBits, size_t bitsSize)
    {
        thrust::exclusive_scan(thrust::device, d_frameStartIndiciesBits, d_frameStartIndiciesBits + bitsSize, d_frameStartIndiciesBits);
    }

}