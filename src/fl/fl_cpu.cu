#include "hip/hip_runtime.h"
#include <cmath>
#include <stdexcept>

#include "fl_cpu.cuh"
#include "../timers/cpu_timer.cuh"

namespace FixedLength
{
    FLCompressed cpuCompress(uint8_t *data, size_t size)
    {
        if (size == 0)
        {
            return FLCompressed{
                .outputBits = nullptr,
                .bitsSize = 0,
                .outputValues = nullptr,
                .valuesSize = 0,
                .inputSize = 0};
        }

        Timers::CpuTimer cpuTimer;

        cpuTimer.start();

        // Allocate bits array
        const size_t framesCount = ceil(size * 1.0 / FRAME_LENGTH);

        uint8_t *outputBits = reinterpret_cast<uint8_t *>(malloc(sizeof(uint8_t) * framesCount));

        if (outputBits == nullptr)
        {
            throw std::runtime_error("Cannot allocate memory\n");
        }

        cpuTimer.end();
        cpuTimer.printResult("Allocate an array on CPU");

        cpuTimer.start();

        // Calculate outputBits
        size_t totalBitsRequired = 0;
        for (size_t f = 0; f < framesCount; f++)
        {
            // We set it to 1 so that when we have 0 (which in our computation returns requiredBits = 0)
            // we don't have to manually adjust it every time that it actually need 1 bit.
            uint8_t minBits = 1;
            for (size_t i = 0; i < FRAME_LENGTH && f * FRAME_LENGTH + i < size; i++)
            {
                uint8_t leadingZeroes = countLeadingZeroes(data[f * FRAME_LENGTH + i]);
                uint8_t requiredBits = 8 - leadingZeroes;
                minBits = max(minBits, requiredBits);
            }
            outputBits[f] = minBits;
            totalBitsRequired += minBits * min(FRAME_LENGTH, size - FRAME_LENGTH * f);
        }

        // Allocate values array
        const size_t valuesSize = ceil(totalBitsRequired * 1.0 / 8);
        uint8_t *outputValues = reinterpret_cast<uint8_t *>(malloc(sizeof(uint8_t) * valuesSize));
        memset(outputValues, 0, sizeof(uint8_t) * valuesSize);

        cpuTimer.end();
        cpuTimer.printResult("Calculate required bits + allocate output array");

        cpuTimer.start();

        // Compression
        size_t usedBits = 0;
        for (size_t f = 0; f < framesCount; f++)
        {
            uint8_t requiredBits = outputBits[f];
            for (size_t i = 0; i < FRAME_LENGTH && f * FRAME_LENGTH + i < size; i++)
            {
                uint8_t value = data[f * FRAME_LENGTH + i];
                size_t outputId = usedBits / 8;
                uint8_t outputOffset = usedBits % 8;
                // Encode value
                uint8_t encodedValue = value << outputOffset;
                outputValues[outputId] |= encodedValue;

                // If it overflows encode the overflowed part on next byte
                if (outputOffset + requiredBits > 8)
                {
                    uint8_t overflowValue = value >> (8 - outputOffset);
                    outputValues[outputId + 1] |= overflowValue;
                }
                usedBits += requiredBits;
            }
        }

        cpuTimer.end();
        cpuTimer.printResult("Compression");

        return FLCompressed{
            .outputBits = outputBits,
            .bitsSize = framesCount,
            .outputValues = outputValues,
            .valuesSize = valuesSize,
            .inputSize = size};
    }

    FLDecompressed cpuDecompress(size_t outputSize, uint8_t *bits, size_t bitsSize, uint8_t *values, size_t valuesSize)
    {
        if (valuesSize == 0 || bitsSize == 0)
        {
            return FLDecompressed{
                .data = nullptr,
                .size = 0};
        }

        Timers::CpuTimer cpuTimer;

        cpuTimer.start();

        // Allocate needed data
        uint8_t *data = reinterpret_cast<uint8_t *>(malloc(sizeof(uint8_t) * outputSize));
        if (data == nullptr)
        {
            throw std::runtime_error("Cannot allocate memory\n");
        }

        cpuTimer.end();
        cpuTimer.printResult("Allocate an array on CPU");

        cpuTimer.start();

        // Decompression
        size_t consumedBits = 0;
        for (size_t f = 0; f < bitsSize; f++)
        {
            uint8_t usedBits = bits[f];
            for (size_t i = 0; i < FRAME_LENGTH && (f * FRAME_LENGTH + i) < outputSize; i++)
            {
                size_t outputId = f * FRAME_LENGTH + i;
                size_t inputId = consumedBits / 8;
                uint8_t inputOffset = consumedBits % 8;
                uint8_t mask = (1 << usedBits) - 1;

                // Decode value
                uint8_t decodedValue = (values[inputId] >> inputOffset) & mask;
                // If there was overflow when encoding we must take the other part of the number
                if (inputOffset + usedBits > 8)
                {
                    uint8_t overflowBits = inputOffset + usedBits - 8;
                    uint8_t overflowMask = (1 << overflowBits) - 1;
                    uint8_t overflowValue = (values[inputId + 1] & overflowMask) << (usedBits - overflowBits);
                    decodedValue |= overflowValue;
                }

                data[outputId] = decodedValue;
                consumedBits += usedBits;
            }
        }

        cpuTimer.end();
        cpuTimer.printResult("Decompression");

        return FLDecompressed{
            .data = data,
            .size = outputSize};
    }

    uint8_t countLeadingZeroes(uint8_t value)
    {
        if (value == 0)
        {
            return 8;
        }
        uint8_t count = 0;
        uint8_t mask = 1 << 7;
        while (!(value & mask))
        {
            count++;
            value <<= 1;
        }
        return count;
    }

} // FixedLength